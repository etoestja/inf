//matrix [столбец][строка]
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void kernel (unsigned N, unsigned* a, unsigned* b, unsigned* r){
	unsigned i = blockIdx.x * blockDim.x + threadIdx.x; //Строка
	unsigned j = blockIdx.y * blockDim.y + threadIdx.y; //Столбец
	if (i<N && j<N){
	//    for (unsigned iterator=0; iterator<N ; ++iterator){
	//   	r[i*N+j] += b[iterator*N+j] * a[i*N+iterator];
	//    } 
//		r[i * N + j] = b[i * N + j] + a[i * N + j];
        r[i] += b[i];
        r[i] -= b[i];
	}
}

extern "C"{
	
	void MatrixMul(unsigned size,unsigned *a, unsigned *b, unsigned *r){
		//printf("\ncalling kernel\n");
		//printmat(r);
		dim3 threadsPerBlock(size, size);
	    //dim3 numBlocks(size / threadsPerBlock.x, size / threadsPerBlock.y);
	    kernel<<<1, threadsPerBlock>>>(size,a,b,r);
	    // printf("kernel done");
	}
}
