
#include <hip/hip_runtime.h>
#include <stdio.h>

// Variables
float* h_A;
float* h_B;
float* h_C;
float* d_A;
float* d_B;
float* d_C;

// Functions
void Cleanup(bool);
void RandomInit(float*, int);
void ParseArguments(int, char**);

// Device code
__global__ void VecAdd(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] - B[i];
}

// Host code
int main(int argc, char** argv)
{
    int N;
    scanf("%d", &N);

    size_t size = N * sizeof(float);

    // Allocate input vectors h_A and h_B in host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    hipError_t error;
    
    // Initialize input vectors
    for(int i = 0; i < N; i++)
    {
        scanf("%f", h_A + i);
        scanf("%f", h_B + i);
    }


    // Allocate vectors in device memory
    error = hipMalloc((void**)&d_A, size);
    error = hipMalloc((void**)&d_B, size);
    error = hipMalloc((void**)&d_C, size);

    // Copy vectors from host memory to device memory
    error = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    error = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    
    error = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    int i;
    for (i = 0; i < N; ++i)
        printf("%f+%f=%f\n", h_A[i], h_B[i], h_C[i]);
}
